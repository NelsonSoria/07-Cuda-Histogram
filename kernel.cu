#include <cstdint>
#include <hip/hip_runtime.h>

__global__ void histogram_kernel(const uint8_t* img, int img_size, int* hist) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < img_size) {
        atomicAdd(&hist[img[idx]], 1);
    }
}

void histogramCUDA(const uint8_t* img, int img_size, int* hist_host) {
    uint8_t* d_img;
    int* d_hist;
    hipMalloc(&d_img, img_size);
    hipMalloc(&d_hist, 256 * sizeof(int));
    hipMemcpy(d_img, img, img_size, hipMemcpyHostToDevice);
    hipMemset(d_hist, 0, 256 * sizeof(int));

    int blockSize = 256;
    int numBlocks = (img_size + blockSize - 1) / blockSize;
    histogram_kernel<<<numBlocks, blockSize>>>(d_img, img_size, d_hist);

    hipMemcpy(hist_host, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_img);
    hipFree(d_hist);
}